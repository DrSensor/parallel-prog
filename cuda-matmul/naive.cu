#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixMulKernel(double *A, double *B, double *C, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < N && col < N) {
    double sum = 0.0;
    for (int k = 0; k < N; k++) {
      sum += A[row * N + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
  }
}

int main() {
  int N = 512;
  size_t bytes = N * N * sizeof(double);

  double *h_A = (double *)malloc(bytes);
  double *h_B = (double *)malloc(bytes);
  double *h_C = (double *)malloc(bytes);

  // Initialize matrices
  for (int i = 0; i < N * N; i++) {
    h_A[i] = 1.0;
    h_B[i] = 2.0;
    h_C[i] = 0.0;
  }

  double *d_A, *d_B, *d_C;
  hipMalloc(&d_A, bytes);
  hipMalloc(&d_B, bytes);
  hipMalloc(&d_C, bytes);

  hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

  matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
